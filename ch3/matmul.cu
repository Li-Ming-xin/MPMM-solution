#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

__global__ void matmul(const float *A, const float *B, float *C, const int M,
                       const int N, const int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M && col < N) {
    float acc = 0.f;
    for (int k = 0; k < K; ++k) {
      acc += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = acc;
  }
}

void matmul_stub(const float *h_A, const float *h_B, float *h_C, const int M,
                 const int N, const int K, const int Asize, const int Bsize,
                 const int Csize) {
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, Asize);
  hipMalloc(&d_B, Bsize);
  hipMalloc(&d_C, Csize);

  hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);

  printf("gpu begin...\n");

  dim3 dimGrid(3, 4), dimBlock(8, 8);
  matmul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);

  printf("gpu end\n");

  hipMemcpy(h_C, d_C, Csize, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

__global__ void matmul_row(const float *A, const float *B, float *C,
                           const int M, const int N, const int K) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M) {
    for (int col = 0; col < N; ++col) {
      float acc = 0.f;
      for (int k = 0; k < K; ++k) {
        acc += A[row * K + k] * B[k * N + col];
      }
      C[row * N + col] = acc;
    }
  }
}

void matmul_row_stub(const float *h_A, const float *h_B, float *h_C,
                     const int M, const int N, const int K, const int Asize,
                     const int Bsize, const int Csize) {
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, Asize);
  hipMalloc(&d_B, Bsize);
  hipMalloc(&d_C, Csize);

  hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);

  printf("gpu begin...\n");

  matmul_row<<<4, 8>>>(d_A, d_B, d_C, M, N, K);

  printf("gpu end\n");

  hipMemcpy(h_C, d_C, Csize, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

__global__ void matmul_col(const float *A, const float *B, float *C,
                           const int M, const int N, const int K) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < N) {
    for (int row = 0; row < M; ++row) {
      float acc = 0.f;
      for (int k = 0; k < K; ++k) {
        acc += A[row * K + k] * B[k * N + col];
      }
      C[row * N + col] = acc;
    }
  }
}

void matmul_col_stub(const float *h_A, const float *h_B, float *h_C,
                     const int M, const int N, const int K, const int Asize,
                     const int Bsize, const int Csize) {
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, Asize);
  hipMalloc(&d_B, Bsize);
  hipMalloc(&d_C, Csize);

  hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);

  printf("gpu begin...\n");

  matmul_col<<<3, 8>>>(d_A, d_B, d_C, M, N, K);

  printf("gpu end\n");

  hipMemcpy(h_C, d_C, Csize, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

void matmul_naive(const float *A, const float *B, float *C, const int M,
                  const int N, const int K) {
  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      float acc = 0.f;
      for (int k = 0; k < K; ++k) {
        acc += A[m * K + k] * B[k * N + n];
      }
      C[m * N + n] = acc;
    }
  }
}

int main() {
  int M = 32, N = 24, K = 56;

  size_t Asize = M * K * sizeof(float);
  size_t Bsize = K * N * sizeof(float);
  size_t Csize = M * N * sizeof(float);

  float *h_A = (float *)malloc(Asize);
  float *h_B = (float *)malloc(Bsize);
  float *h_C = (float *)malloc(Csize);
  float *h_C_naive = (float *)malloc(Csize);

  for (int m = 0; m < M; ++m) {
    for (int k = 0; k < K; ++k) {
      h_A[m * K + k] = m * K + k;
    }
  }
  for (int k = 0; k < K; ++k) {
    for (int n = 0; n < N; ++n) {
      h_B[k * N + n] = k * N + n;
    }
  }

  // matmul_stub(h_A, h_B, h_C, M, N, K, Asize, Bsize, Csize);
  // matmul_row_stub(h_A, h_B, h_C, M, N, K, Asize, Bsize, Csize);
  matmul_col_stub(h_A, h_B, h_C, M, N, K, Asize, Bsize, Csize);

  printf("cpu begin...\n");

  matmul_naive(h_A, h_B, h_C_naive, M, N, K);
  printf("cpu end\n");

  free(h_A);
  free(h_B);

  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      if (fabsf(h_C[m * N + n] - h_C_naive[m * N + n]) > 1e-5) {
        printf("wrong\n\tm: %d, n: %d, host value: %f, device value: %f\n", m,
               n, h_C_naive[m * N + n], h_C[m * N + n]);
        free(h_C);
        exit(1);
      }
    }
  }

  printf("done\n");
  free(h_C);

  return 0;
}
